#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <math.h>
#include <>
#include <stdio.h>
#include <SFML/Graphics.hpp>
#include <imgui-SFML.h>
#include <imgui.h>

#define IMAGE_X 1920
#define IMAGE_Y 1080

__device__ __host__ float length(const float3& v)
{
	return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ __host__ float3 operator-(const float3& a, const float3& b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ __host__ float3 operator+(const float3& a, const float3& b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ __host__ float3 operator*(const float3& a, const float3& b)
{
	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__device__ __host__ float3 operator*(const float3& a, float b)
{
	return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ __host__ float3 operator/(const float3& a, float b)
{
	return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ __host__ float3 unitVector(const float3& v)
{
	return v / length(v);
}


__interface IHittable
{
	//Get distance from query point to the surface of the object
	__device__ __host__ float SDF(const float3& pos) const;

};

class Sphere : public IHittable
{
public:
	float3 center;
	float radius;

	__device__ __host__ Sphere(float3 center, float radius) : center(center), radius(radius) {}

	__device__ __host__ float SDF(const float3& pos) const override
	{
		return length(pos - center) - radius;
	}


};

__global__ void renderSphereOnly(Sphere sphereIn, int3* image, float3 cameraLookAt, float3 cameraLocation) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= IMAGE_X || y >= IMAGE_Y) return;

	float u = float(x) / IMAGE_X;
	float v = float(y) / IMAGE_Y;

	float3 cameraRayDir = unitVector(cameraLookAt - cameraLocation);
	//Displace the ray based on the pixel position
	float aspectRatio = float(IMAGE_X) / float(IMAGE_Y);

	cameraRayDir.x = cameraRayDir.x + (2 * u - 1) * aspectRatio * 0.1;
	cameraRayDir.y = cameraRayDir.y + (2 * v - 1) * 0.1;

	//Normalize the ray direction (NECESSARY OTHERWISE IT CAUSES ALL KINDS OF WEIRD DISTORTIONS)
	cameraRayDir = unitVector(cameraRayDir);


	float3 pos = cameraLocation;
	float t = 0;
	float dist = sphereIn.SDF(pos);



	int maxIter = 1000;
	
	for (int i = 0; i < maxIter; i++) {
		//printf("Distance to sphere: %f\n", dist);
		if (x==250 && y==250) printf("Distance to sphere: %f\n", dist);
			if (dist < 0.001) {
				image[y * IMAGE_X + x] = make_int3(255, 255, 255);
				return;
			} else if (t > 100) {
				image[y * IMAGE_X + x] = make_int3(0, 0, 0);
				return;
			}
			t += dist;
			pos = cameraLocation + cameraRayDir * t;
			dist = sphereIn.SDF(pos);
			
	}
}

void writeImageToFile(int3* image, int width, int height) {
	FILE* file = fopen("image.ppm", "w");
	fprintf(file, "P3\n%d %d\n%d\n", width, height, 255);
	for (int i = 0; i < width * height; i++) {
		fprintf(file, "%d %d %d ", image[i].x, image[i].y, image[i].z);
	}
	fclose(file);
}




int main() {
	int3* image = new int3[IMAGE_X * IMAGE_Y];
	int3* d_image;

	hipMalloc(&d_image, IMAGE_X * IMAGE_Y * sizeof(int3));
	Sphere sphere(make_float3(0, 1, -1.2), 1.19);

	//Camera
	float3 cameraLocation = make_float3(2.98, -0.8, 1.3);
	float3 cameraLookAt = make_float3(-0.226, 0.972, 0.122);
	

	

	

	//Display image
	sf::RenderWindow window(sf::VideoMode(IMAGE_X, IMAGE_Y), "Raymarching");
	ImGui::SFML::Init(window);
	sf::Texture texture;
	texture.create(IMAGE_X, IMAGE_Y);
	sf::Sprite sprite(texture);
	sf::Uint8* pixels = new sf::Uint8[IMAGE_X * IMAGE_Y * 4];
	sf::Clock deltaClock;



while (window.isOpen()) {
		sf::Event event;
		while (window.pollEvent(event)) {
			ImGui::SFML::ProcessEvent(window, event);
			if (event.type == sf::Event::Closed) {
				window.close();
			}
		}
		ImGui::SFML::Update(window, deltaClock.restart());


		ImGui::Begin("Hello, world!");

		//Set sphere center
		ImGui::DragFloat3("Center", &sphere.center.x, 0.01f, -10.0f, 10.0f);
		ImGui::DragFloat("Radius", &sphere.radius, 0.01f, 0.0f, 10.0f);

		//Set camera location
		ImGui::DragFloat3("Camera Location: ", &cameraLocation.x, 0.01f, -100.f, 100.0f);
		ImGui::DragFloat3("Camera Look At: ", &cameraLookAt.x, 0.01f, -10.0f, 10.0f);
		if (ImGui::Button("Render")) {
			dim3 blockSize(16, 16);
			dim3 numBlocks((IMAGE_X + blockSize.x - 1) / blockSize.x, (IMAGE_Y + blockSize.y - 1) / blockSize.y);
			renderSphereOnly << <numBlocks, blockSize >> > (sphere, d_image, cameraLookAt, cameraLocation);
			hipMemcpy(image, d_image, IMAGE_X * IMAGE_Y * sizeof(int3), hipMemcpyDeviceToHost);
			for (int i = 0; i < IMAGE_X * IMAGE_Y; i++) {
				pixels[i * 4] = image[i].x;
				pixels[i * 4 + 1] = image[i].y;
				pixels[i * 4 + 2] = image[i].z;
				pixels[i * 4 + 3] = 255;
			}
			printf("Rendered!");
			texture.update(pixels);
		}
		
		ImGui::End();

		

		window.clear();
		window.draw(sprite);
		ImGui::SFML::Render(window);
		window.display();
	}


	writeImageToFile(image, IMAGE_X, IMAGE_Y);

	//SDF TEST
	float3 pos = make_float3(0, 1, -1.2);
	float dist = sphere.SDF(pos);
	printf("Distance to sphere: %f\n", dist);

	
}